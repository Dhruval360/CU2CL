//This case is used to demonstrate a case where CU2CL
// can generate a multiple declaration of anew cl_mem variable
// that results when translating a hipHostAlloc'd pointer

//This occurs as the variable is generated at global scope, rather than
// inheriting the scope of the original CUDA pointer

//A correct translation would move the global declaration of __cu2cl_Mem_h_odata
// into the two individual functions where h_odata is locally-declared
#include "../../test_utils.hpp"

int combined;

void configHostAlloc(int testCount, int writeCombined) {
	count = testCount;
	combined = writeCombined;
}

//Test host-allocing two buffers, and transfering data through a device buffer
void testHostAlloc1() {
	unsigned char * h_idata = NULL;
	unsigned char * h_odata = NULL;
	unsigned char * d_idata;
#if CUDART_VERSION >= 2020
	hipHostAlloc((void**)&h_idata, count*sizeof(unsigned char), combined ? hipHostMallocWriteCombined : 0);
	hipHostAlloc((void**)&h_odata, count*sizeof(unsigned char), combined ? hipHostMallocWriteCombined : 0);
#else
	hipHostMalloc((void**)&h_idata, count*sizeof(unsigned char));
	hipHostMalloc((void**)&h_odata, count*sizeof(unsigned char));
#endif

	//init some host data in h_idata
	randomFillBuff<unsigned char>(h_idata, 0, 255, count);

	hipMalloc((void**) &d_idata, count*sizeof(unsigned char));

	hipMemcpy(d_idata, h_idata, count*sizeof(unsigned char), hipMemcpyHostToDevice);

	hipMemcpy(h_odata, d_idata, count*sizeof(unsigned char), hipMemcpyDeviceToHost);

	//check data integrity
	checkBufBufExact<unsigned char>(h_odata, h_idata, count);

	hipHostFree(h_idata);
	hipHostFree(h_odata);
	hipFree(d_idata);
}

void testHostAlloc2() {
	unsigned char * h_odata = NULL;
	unsigned char * d_idata;
#if CUDART_VERSION >= 2020
	hipHostAlloc((void**)&h_odata, count*sizeof(unsigned char), combined ? hipHostMallocWriteCombined : 0);
#else
	hipHostMalloc((void**)&h_odata, count*sizeof(unsigned char));
#endif

	//No need to init data, the validity is established by testHostAlloc1
	hipMalloc((void**)&d_idata, count*sizeof(unsigned char));
	hipMemcpy(d_idata, h_odata, count*sizeof(unsigned char), hipMemcpyHostToDevice);

	hipHostFree(h_odata);
	hipFree(d_idata);
}
